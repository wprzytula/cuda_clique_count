#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "common/errors.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <memory>
#include <vector>
#include <utility>
#include <charconv>
#include <algorithm>
#include <numeric>
#include <cassert>


// https://stackoverflow.com/a/18968893
// Prior to the kernel define these helper functions and device variable:
__device__ volatile int sem = 0;

__device__ void acquire_semaphore(volatile int *lock){
    while (atomicCAS((int *)lock, 0, 1) != 0);
}

__device__ void release_semaphore(volatile int *lock){
    *lock = 0;
    __threadfence();
}

// #define PRINT

#ifdef PRINT
constexpr bool const debug = true;
#else
constexpr bool const debug = false;
#endif

#define MAX_K 12
#define MAX_DEG 1024
#define BLOCK_SIZE 32
// #define BLOCK_SIZE 2
#define NUM_BLOCKS 64
// #define NUM_BLOCKS 1

namespace cpu { namespace {

// Graph traversal for graph orientation method
// 1 𝑛𝑢𝑚𝐶𝑙𝑖𝑞𝑢𝑒𝑠 = 0
// 2 procedure 𝑡𝑟𝑎𝑣𝑒𝑟𝑠𝑒𝑆𝑢𝑏𝑡𝑟𝑒𝑒 (𝐺, 𝑘, ℓ, 𝐼 ) : (G: Graph, k: clique_size, l: current_level, I: set_of_vertices)
// 3 for 𝑣 ∈ 𝐼
// 4    𝐼 ′ = 𝐼 ∩ 𝐴𝑑𝑗_𝐺 (𝑣)
// 5    if ℓ + 1 == 𝑘
// 6        𝑛𝑢𝑚𝐶𝑙𝑖𝑞𝑢𝑒𝑠 + = |𝐼 ′ |
// 7    else if |𝐼 ′ | > 0
// 8        𝑡𝑟𝑎𝑣𝑒𝑟𝑠𝑒𝑆𝑢𝑏𝑡𝑟𝑒𝑒 (𝐺, 𝑘, ℓ + 1, 𝐼 ′ )

    using Edge = std::pair<int, int>;

    // TODO: remap vertices so that they are consecutive natural numbers

    int find_max_vertex(std::vector<Edge> const& edges) {
        int max_vertex = 0;
        for (auto const [v1, v2]: edges) {
            if (v1 > max_vertex) {
                max_vertex = v1;
            }
            if (v2 > max_vertex) {
                max_vertex = v2;
            }
        }
        // std::cerr << "Max vertex found: " << max_vertex << std::endl;
        return max_vertex;
    }

    std::vector<int> compute_degs(std::vector<Edge> const& edges, int max_vertex) {
        std::vector<int> deg;
        deg.resize(max_vertex + 1);

        for (auto const [v1, v2]: edges) {
            ++deg[v1];
            ++deg[v2];
        }

        return deg;
    }

    void orient_graph(std::vector<Edge>& edges, std::vector<int> const& deg) {
        for (Edge& edge: edges) {
            auto [v1, v2] = edge;
            int const deg_v1 = deg[v1];
            int const deg_v2 = deg[v2];
            if (deg_v1 > deg_v2 || (deg_v1 == deg_v2 && v1 > v2)) {
                // Revert the edge
                edge.first = v2;
                edge.second = v1;
            }
        }
    }

    struct CSR {
        std::vector<int> col_idx;
        std::vector<int> row_ptr;
        int max_v;
        int n;

        CSR(std::vector<Edge> const& edges) : max_v{find_max_vertex(edges)}, n{max_v + 1} {
            assert(std::is_sorted(edges.cbegin(), edges.cend()));

            col_idx.resize(edges.size());
            row_ptr.resize(n + 1);

            int col_i = 0;
            for (int row = 0; row <= max_v; ++row) {
                row_ptr[row] = col_i;
                while (col_i < edges.size() && edges[col_i].first == row) {
                    // std::cerr << "Col_i: " << col_i << std::endl;
                    col_idx[col_i] = edges[col_i].second;
                    ++col_i;
                }
            }
            row_ptr[n] = col_idx.size();
        }
    };

    std::ostream& operator<<(std::ostream &os, CSR const& csr) {
        os << "Col_idx: [ ";
        for (int col: csr.col_idx) {
            os << col << ", ";
        }
        os << " ]\n";
        os << "Row_ptr: [ ";
        for (int row: csr.row_ptr) {
            os << row << ", ";
        }
        os << "]\n";

        return os;
    }

    Edge parse_edge(std::string const& buf) {
        char const* ptr = buf.data();
        int v1, v2;
        auto res1 = std::from_chars(ptr, ptr + buf.size(), v1);
        if (res1.ec != std::errc()) {
            std::cerr << "Error while parsing first vertex int!\n";
            std::cerr << "(problematic line: " << ptr << ")\n";
            exit(EXIT_FAILURE);
        }
        ptr = res1.ptr;
        while (std::isspace(*ptr)) ++ptr;

        auto res2 = std::from_chars(ptr, buf.data() + buf.size(), v2);
        if (res2.ec != std::errc()) {
            std::cerr << "Error while parsing second vertex int!\n";
            std::cerr << "(problematic line: " << ptr << ")\n";
            exit(EXIT_FAILURE);
        }
        return {v1, v2};
    }

    struct InducedSubgraph {
        std::vector<int> const mapping;
        std::vector<std::vector<int>> const adjacency_matrix;
private:
        InducedSubgraph(std::vector<int> mapping, std::vector<std::vector<int>> adjacency_matrix)
            : mapping{std::move(mapping)}, adjacency_matrix{std::move(adjacency_matrix)} {}

public:
        static InducedSubgraph extract(CSR const& graph, int vertex) {
            int const i = vertex;

/* Build subgraph mapping: new_vertex [0..1024] -> old_vertex [0..|V|] */
            std::vector<int> subgraph_mapping;
            int const start = graph.row_ptr[i];
            int const end = graph.row_ptr[i + 1];
            for (int j = start; j < end; ++j) {
                // put neighbours in mapping.
                int const neighbour = graph.col_idx[j];
                subgraph_mapping.push_back(neighbour);
            }

/* Build adjacency matrix  */
            std::vector<std::vector<int>> adjacency_matrix;

            // It has k rows, where k = |induced subgraph vertices|
            adjacency_matrix.resize(subgraph_mapping.size());

            auto old = [&subgraph_mapping](int new_v){/* std::cout << "old(" << new_v << ")\n";  */return subgraph_mapping[new_v];};
            auto neigh = [&graph](int col_i){return graph.col_idx[col_i];};

            // For each row
            for (int i = 0; i < subgraph_mapping.size(); ++i) {
                // Retrieve old id of the vertex
                int const old_v1 = subgraph_mapping[i];
                // std::cout << "Row with new id: " << i << ", old id: " << old_v1 << "\n";

                // Operate on this row
                auto& row = adjacency_matrix[i];
                // Resize it to k
                row.resize(subgraph_mapping.size());

                int csr_idx = graph.row_ptr[old_v1];
                int const csr_idx_end = graph.row_ptr[old_v1 + 1];

                // For each cell in this row
                for (int adj_idx = 0; adj_idx < subgraph_mapping.size(); ++adj_idx) {
                    // std::cout << "Incremented adj_idx to " << adj_idx << ", now points to " << old(adj_idx) << "\n";

                    if (csr_idx >= csr_idx_end) {
                            // std::cout << "csr_idx went out of bounds.\n";
                            goto end_row;
                    }

                    while (neigh(csr_idx) < old(adj_idx)) {
                        // std::cout << "Incremented csr_idx to " << csr_idx << "\n";
                        ++csr_idx;
                        if (csr_idx >= csr_idx_end) {
                            // std::cout << "csr_idx went out of bounds.\n";
                            goto end_row;
                        }
                        // std::cout << "csr_idx now points to " << neigh(csr_idx) << "\n";
                    }

                    // printf("Deciding edge between %d and %d based on value in csr_idx under %d: %d\n",
                    //      old_v1, old(adj_idx), csr_idx, neigh(csr_idx));
                    row[adj_idx] = neigh(csr_idx) == old(adj_idx);
end_row:            ;
                }
            }
            return InducedSubgraph{subgraph_mapping, adjacency_matrix};
        }
        InducedSubgraph operator=(InducedSubgraph const&) = delete;
    };
    std::ostream& operator<<(std::ostream &os, InducedSubgraph const& subgraph) {
        os << "Subgraph mapping: [ ";
        for (int old_v: subgraph.mapping) {
            os << old_v << " ";
        }
        os << "]\n";
        os << "Adjacency matrix:\n";
        os << "  ";
        for (int old_v: subgraph.mapping) {
            os << old_v << " ";
        }
        os << "\n";
        for (auto const& row: subgraph.adjacency_matrix) {
            os << "[";
            for (bool exists: row) {
                os << ' ' << (exists ? 'x' : ' ');
            }
            os << " ]\n";
        }

        return os;
    }
}} // namespace


struct CSR {
    int vs;
    int* row_ptr;
    int row_len;
    int* col_idx;
    int col_len;
};

struct InducedSubgraph {
    int len;
    int mapping[MAX_DEG]; // len
    int adjacency_matrix[MAX_DEG * MAX_DEG]; // len * len

    __device__ void extract(CSR const& graph, int const v) {
        int const tid = threadIdx.x;
        int const start = graph.row_ptr[v];
        int const end = graph.row_ptr[v + 1];

        if (tid == 0) {
            len = end - start;
        }
        __syncthreads();

/* Build subgraph mapping: new_vertex [0..1024] -> old_vertex [0..|V|] */
        for (int j = tid; start + j < end; j += blockDim.x) {
            // put neighbours in mapping.
            int const neighbour = graph.col_idx[start + j];
            mapping[j] = neighbour;
        }

        __syncthreads();

/* Build adjacency matrix  */
        // It has k rows, where k = |induced subgraph vertices|
        auto const& mapping = this->mapping;
        auto old = [&mapping](int new_v){/* std::cout << "old(" << new_v << ")\n";  */return mapping[new_v];};
        auto neigh = [&graph](int col_i){return graph.col_idx[col_i];};

        // For each row
        for (int i = tid; i < len; i += blockDim.x) {
            // Retrieve old id of the vertex
            int const old_v1 = mapping[i];

            // Operate on this row
            auto *const row = adjacency_matrix + i * len;

            // Clear the row after previous subgraph (zeros are assumed in the algorithm)
            for (int j = 0; j < len; ++j) {
                row[j] = false;
            }

            int csr_idx = graph.row_ptr[old_v1];
            int const csr_idx_end = graph.row_ptr[old_v1 + 1];

            // For each cell in this row
            for (int adj_idx = 0; adj_idx < len; ++adj_idx) {
                // std::cout << "Incremented adj_idx to " << adj_idx << ", now points to " << old(adj_idx) << "\n";

                if (csr_idx >= csr_idx_end) {
                    // std::cout << "csr_idx went out of bounds.\n";
                    goto end_row;
                }

                while (neigh(csr_idx) < old(adj_idx)) {
                    // std::cout << "Incremented csr_idx to " << csr_idx << "\n";
                    ++csr_idx;
                    if (csr_idx >= csr_idx_end) {
                        // std::cout << "csr_idx went out of bounds.\n";
                        goto end_row;
                    }
                    // std::cout << "csr_idx now points to " << neigh(csr_idx) << "\n";
                }

                // printf("Deciding edge between %d and %d based on value in csr_idx under %d: %d\n",
                //      old_v1, old(adj_idx), csr_idx, neigh(csr_idx));
                row[adj_idx] = neigh(csr_idx) == old(adj_idx);
end_row:            ;
            }
        }
    }
};

__device__ void print_subgraph(InducedSubgraph const& subgraph) {
        printf("Subgraph mapping: [ ");
        for (int i = 0; i < subgraph.len; ++i) {
            int const old_v = subgraph.mapping[i];
            printf("%i ", old_v);
        }
        printf("]\n");
        printf("Adjacency matrix:\n");
        printf("  ");
        for (int i = 0; i < subgraph.len; ++i) {
            int const old_v = subgraph.mapping[i];
            printf("%i ", old_v);
        }
        printf("\n");
        for (int i = 0; i < subgraph.len; ++i) {
            printf("[");
            for (int j = 0; j < subgraph.len; ++j) {
                bool exists = subgraph.adjacency_matrix[i * subgraph.len + j];
                printf(" %c", exists ? 'x' : ' ');
            }
            printf(" ]\n");
        }
    }

struct Stack {
    // VertexSet
    bool *vertices; // 2-level array [[true, true, false], [false, false, false]]

    bool* done;

    int* level; // len: stack_entries_num
};

struct Data {
    int const k;
    int* next_vertex;
    Stack stacks[NUM_BLOCKS];
    CSR csr;
    InducedSubgraph* subgraphs;

    Data(cpu::CSR const& edges, int const k) : k{k} {
        csr.vs = edges.n;

        csr.row_len = edges.row_ptr.size();
        csr.row_ptr = nullptr;
        HANDLE_ERROR(hipMalloc(&csr.row_ptr, edges.row_ptr.size() * sizeof(int)));
        HANDLE_ERROR(hipMemcpy(
            csr.row_ptr,
            edges.row_ptr.data(),
            edges.row_ptr.size() * sizeof(int),
            hipMemcpyHostToDevice)
        );

        csr.col_len = edges.col_idx.size();
        csr.col_idx = nullptr;
        HANDLE_ERROR(hipMalloc(&csr.col_idx, edges.col_idx.size() * sizeof(int)));
        HANDLE_ERROR(hipMemcpy(
            csr.col_idx,
            edges.col_idx.data(),
            edges.col_idx.size() * sizeof(int),
            hipMemcpyHostToDevice)
        );

        HANDLE_ERROR(hipMalloc(&subgraphs, NUM_BLOCKS * sizeof(InducedSubgraph)));

        // Initialise stacks
        int const max_entries = k * MAX_DEG;
        for (int i = 0; i < NUM_BLOCKS; ++i) {
            Stack& stack = stacks[i];
            HANDLE_ERROR(hipMalloc(&stack.vertices, max_entries * MAX_DEG * sizeof(*stack.vertices)));
            HANDLE_ERROR(hipMemset(stack.vertices, /* -1 */1, MAX_DEG * sizeof(*stack.vertices))); // first stack entry

            HANDLE_ERROR(hipMalloc(&stack.level, max_entries * sizeof(*stack.level)));
            HANDLE_ERROR(hipMalloc(&stack.done, max_entries * sizeof(*stack.done)));
        }

        HANDLE_ERROR(hipMalloc(&next_vertex, sizeof(*next_vertex)));
        HANDLE_ERROR(hipMemset(next_vertex, 0, sizeof(*next_vertex)));
    }
};

__device__ void intersect_adjacent(InducedSubgraph const& subgraph, bool const* vertex_set, int vertex, bool* out_vertex_set) {
        auto const* row = subgraph.adjacency_matrix + vertex * subgraph.len;

        for (int i = threadIdx.x; i < subgraph.len; i += blockDim.x) {
            if (debug) printf("Block %i, Thread %i: I'm intersecting %i-th vertex: vertex_set[%i]=%i, row[%i]=%i\n",
                    blockIdx.x, threadIdx.x, i, i, vertex_set[i], i, row[i]);
            out_vertex_set[i] = vertex_set[i] && row[i]; // set vertex as in or out of set
        }
    }

__device__ bool vertex_set_nonempty(bool const* set, int const len) {
    int const tid = threadIdx.x;

    // if (tid == 0 && debug) {
    //     printf("Reduction!\n");
    //     printf("set[ ");
    //     for (int i = 0; i < len; ++i) {
    //         printf("%p: %i\n", set + i, set[i]);
    //     }
    //     printf("]\n");
    // }
    // __syncthreads();

    __shared__ bool nonempty[BLOCK_SIZE];
    nonempty[tid] = 0;
    for (int i = tid; i < len; i += blockDim.x) {
        // if (debug) printf("Thread %i: nonempty[%i] to %p: %i\n", tid, tid, set + i, i < len ? set[i] : 0);
        nonempty[tid] |= i < len ? set[i] : 0;
    }

    // __syncthreads();
    // if (tid == 0 && debug) {
    //     printf("nonempty([ ");
    //     for (int i = 0; i < BLOCK_SIZE && i < len; ++i) {
    //         printf("%i ", nonempty[i]);
    //     }
    //     printf("])\n");
    // }

    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (tid < i) {
            // printf("Thread %i: reached reduction step i=%i; [tid]=%i, [tid+i]=%i\n", tid, i, nonempty[tid], nonempty[tid + i]);
            nonempty[tid] |= nonempty[tid + i];
        }
        __syncthreads();
        i /= 2;
    }
    if (tid == 0 && debug) {
        printf("set_nonempty([ ");
        for (int i = 0; i < len; ++i) {
            printf("%i ", set[i]);
        }
        printf("]) = %i\n", nonempty[0]);
    }
    return nonempty[0];
}

__device__ int acquire_next_vertex(Data const& data) {
    int const thread_id = threadIdx.x;
    __shared__ int chosen_vertex;

    if (thread_id == 0) {
        chosen_vertex = atomicAdd(data.next_vertex, 1);
        // printf("Block %i: Acquired vertex %i.\n", blockIdx.x, chosen_vertex);
    }
    __syncthreads();
    return chosen_vertex;
}

// Graph traversal for graph orientation method
// 1 𝑛𝑢𝑚𝐶𝑙𝑖𝑞𝑢𝑒𝑠 = 0
// 2 procedure 𝑡𝑟𝑎𝑣𝑒𝑟𝑠𝑒𝑆𝑢𝑏𝑡𝑟𝑒𝑒 (𝐺, 𝑘, ℓ, 𝐼 ) : (G: Graph, k: clique_size, l: current_level, I: set_of_vertices)
// 3 for 𝑣 ∈ 𝐼
// 4    𝐼 ′ = 𝐼 ∩ 𝐴𝑑𝑗_𝐺 (𝑣)
// 5    if ℓ + 1 == 𝑘
// 6        𝑛𝑢𝑚𝐶𝑙𝑖𝑞𝑢𝑒𝑠 + = |𝐼 ′ |
// 7    else if |𝐼 ′ | > 0
// 8        𝑡𝑟𝑎𝑣𝑒𝑟𝑠𝑒𝑆𝑢𝑏𝑡𝑟𝑒𝑒 (𝐺, 𝑘, ℓ + 1, 𝐼 ′ )
__global__ void kernel(Data data, int *count) {
    int const block_id = blockIdx.x;
    int const thread_id = threadIdx.x;

    int chosen_vertex;

    Stack& stack = data.stacks[block_id];
    __shared__ int stack_top;

    if (debug && block_id == 0 && thread_id == 0) printf("\n\n----- RUNNING KERNEL!!! ------\n\n");

    __shared__ int cliques[MAX_K];
    // Set counters to zeros.
    for (int i = thread_id; i < data.k; i += blockDim.x) {
        cliques[i] = 0;
    }

    // debug
    // if (thread_id == 0 && debug) {
    //     if (block_id == 0) {
    //         printf("Printing subgraphs.\n\n");
    //         for (int i = 0; i < data.csr.vs; ++i) {
    //             print_subgraph(data.subgraphs[i]);
    //         }
    //         printf("\nBeginning STACK ITERATION.\n\n");
    //     } else {
    //         clock_t start = clock();
    //         clock_t now;
    //         for (;;) {
    //             now = clock();
    //             clock_t cycles = now > start ? now - start : now + (0xffffffff - start);
    //             if (cycles >= 100000000) {
    //                 break;
    //             }
    //         }
    //     }
    // }
    __syncthreads();

    while ((chosen_vertex = acquire_next_vertex(data)) < data.csr.vs) {
        if (debug && thread_id == 0) {
            printf("Block %i has acquired vertex %i\n", block_id, chosen_vertex);
        }

        // Compute InducedSubgraph
        {
            InducedSubgraph& subgraph = data.subgraphs[block_id];
            subgraph.extract(data.csr, chosen_vertex);
            if (debug && thread_id == 0) print_subgraph(subgraph);
        }
        InducedSubgraph const& subgraph = data.subgraphs[block_id];

        // Initialise first stack frame.
        // stack.emplace(VertexSet::full(subgraphs[v].mapping.size()), k, v, 1);
        if (thread_id == 0) {
            stack_top = 0;
            stack.level[0] = 1;
            stack.done[0] = false;
        }
        __syncthreads();
        while (stack_top >= 0) {
            __syncthreads();
            int const current = stack_top;
            if (debug && thread_id == 0) {
                printf("Block %i vertex %i operating on stack entry with idx %i, done? %i\n",
                        block_id, chosen_vertex, current, stack.done[current]);
            }
            if (stack.done[current]) {
                if (thread_id == 0)
                    --stack_top;
                __syncthreads();
                continue;
            }
            for (int v = 0; v < subgraph.len; ++v) {
                __syncthreads();
                if (stack.vertices[MAX_DEG * current + v]) { // entry.vertices.contains(v)
                    // We've found a `level`-level clique.
                    if (thread_id == 0)
                        ++cliques[stack.level[current]];
                    __syncthreads();
                    // Let's explore deeper.
                    if (stack.level[current] + 1 < data.k) { // entry.level + 1 < k
                        if (thread_id == 0)
                            ++stack_top;
                        __syncthreads();
                        bool* new_vertices = stack.vertices + stack_top * MAX_DEG;
                        if (thread_id == 0 && debug) printf("Block %i, Vertex %i: Intersecting with subgraph's vertex %i.\n", block_id, chosen_vertex, v);
                        intersect_adjacent(subgraph, stack.vertices + current * MAX_DEG, v, new_vertices);

                        __syncthreads();

                        if (vertex_set_nonempty(new_vertices, subgraph.len)) {
                            // stack.emplace(new_vertices, entry.level + 1);
                            if (thread_id == 0) {
                                stack.level[stack_top] = stack.level[current] + 1;
                                stack.done[stack_top] = false;
                            }
                        } else {
                            if (thread_id == 0)
                                --stack_top;
                            __syncthreads();
                        }
                    }
                }
            }
            // if (thread_id == 0 && debug)
            //     printf("Vertex %i: Reached __syncthreads() at line %d.\n", chosen_vertex, __LINE__ + 1);
            __syncthreads();
            // if (thread_id == 0 && debug)
            //     printf("Vertex %i: Passed __syncthreads() at line %d.\n", chosen_vertex, __LINE__ - 2);
            if (thread_id == 0) {
                stack.done[current] = true;
                if (current == stack_top) /*leaf reached, go back*/{
                    if (debug) printf("Vertex %i: Reached leaf in entry %i.\n", chosen_vertex, current);
                    --stack_top;
                } else {
                    if (debug) printf("Vertex %i: Finished work over node in entry %i.\n", chosen_vertex, current);
                }
            }
            // if (thread_id == 0 && debug)
            //     printf("Vertex %i: Reached __syncthreads() at line %d.\n", chosen_vertex, __LINE__ + 1);
            __syncthreads();
            // if (thread_id == 0 && debug)
            //     printf("Vertex %i: Passed __syncthreads() at line %d.\n", chosen_vertex, __LINE__ - 2);
        }
        if (thread_id == 0 && debug) {
            printf("Block %i, Vertex %i: Finished stack iteration.\n", block_id, chosen_vertex);
        }
    }

    __syncthreads();

    if (debug) {
        // https://stackoverflow.com/a/18968893
        if (thread_id == 0)
            acquire_semaphore(&sem);
        __syncthreads();
        //begin critical section
        // ... your critical section code goes here
        if (thread_id == 0) {
            printf("Block %i: count: [ ", block_id);
            for (int i = 0; i < data.k; ++i) {
                printf("%i ", cliques[i]);
            }
            printf("]\n");
        }
        //end critical section
        __threadfence(); // not strictly necessary for the lock, but to make any global updates in the critical section visible to other threads in the grid
        __syncthreads();
        if (threadIdx.x == 0)
            release_semaphore(&sem);
        __syncthreads();
    }

    for (int i = thread_id; i < data.k; i += blockDim.x) {
        atomicAdd(&count[i], cliques[i]);
    }

    __syncthreads();
    if (thread_id == 0 && debug) {
        printf("Block %i, Finished!\n", block_id);
    }
}

using cpu::Edge;
using cpu::parse_edge;
using cpu::compute_degs;
using cpu::orient_graph;

static void count_cliques(std::vector<Edge>& edges, std::ofstream& output_file, int k, int max_v) {
    std::sort(edges.begin(), edges.end());
    if (debug) {
        std::cout << "unoriented sorted edges:\n";
        for (auto const [v1, v2]: edges) {
            std::cout << "(" << v1 << ", " << v2 << ")\n";
        }
        std::cout << "max_v=" << max_v << ")\n";
    }

    if (debug) { // debug
        cpu::CSR unoriented_graph{edges};
        std::cout << "unoriented graph:\n";
        std::cout << unoriented_graph << "\n";
    }

    auto degs = compute_degs(edges, max_v);
    orient_graph(edges, degs);
    std::sort(edges.begin(), edges.end());

    if (debug) {
        std::cout << "oriented sorted edges:\n";
        for (auto const [v1, v2]: edges) {
            std::cout << "(" << v1 << ", " << v2 << ")\n";
        }
    }

    cpu::CSR graph{edges};
    if (debug) {
        std::cout << "oriented graph:\n";
        std::cout << graph << "\n";
    }

    if (debug) for (int v = 0; v <= max_v; ++v) {
        auto subgraph = cpu::InducedSubgraph::extract(graph, v);
        std::cout << subgraph << "\n";
    }

    auto cliques_cpu = std::make_unique<int[]>(k);

    { // GPU section
        // input data
        Data data{edges, k};

        // output data
        int *cliques_gpu;

        HANDLE_ERROR(hipMalloc(&cliques_gpu, k * sizeof(int)));
        HANDLE_ERROR(hipMemset(cliques_gpu, 0, k * sizeof(int)));

        hipEvent_t kernel_run, stop;
        hipEventCreate(&kernel_run);
        hipEventCreate(&stop);

        hipEventRecord(kernel_run, 0);

        // RUN KERNEL, RUN!
        kernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(data, cliques_gpu);


        // Get back the output data
        HANDLE_ERROR(hipMemcpy(cliques_cpu.get(),
                cliques_gpu,
                k * sizeof(int),
                hipMemcpyDeviceToHost)
        );

        HANDLE_ERROR(hipFree(cliques_gpu));

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float elapsed_kernel;
        HANDLE_ERROR(hipEventElapsedTime(&elapsed_kernel, kernel_run, stop));
        printf("Elapsed kernel: %.3fms\n", elapsed_kernel);

        hipEventDestroy(kernel_run);
        hipEventDestroy(stop);
    }

    cliques_cpu[0] = max_v + 1;

    std::stringstream s;

    s << "count: [ ";
    output_file << cliques_cpu[0];
    s << cliques_cpu[0];
    for (int i = 1; i < k; ++i) {
        output_file << ' ' << cliques_cpu[i];
        s << ' ' << cliques_cpu[i];
    }
    s << " ]\n";
    // if (debug)
    std::cout << s.str();
}

int main(int argc, char const* argv[]) {
    if (argc != 4) {
        fprintf(stderr, "Bad arg num (expected 4, got %d)\n", argc);
        return EXIT_FAILURE;
    }

    char const* input_filename = argv[1];
    char const* k_str = argv[2];
    char const* output_filename = argv[3];

    std::ifstream input_file{input_filename, std::ios::in};
    if (!input_file.is_open()) {
        std::cerr << "Could not open input file '" << input_filename << "'!\n";
        return EXIT_FAILURE;
    }

    int k;
    try {
        k = std::stoi(k_str);
    } catch (std::invalid_argument&) {
        std::cerr << "Non integer k: " << k_str << "'!\n";
        return EXIT_FAILURE;
    } catch (std::out_of_range&) {
        std::cerr << "k too big for int type: " << k_str << "'!\n";
        return EXIT_FAILURE;
    }

    std::ofstream output_file{output_filename, std::ios::out};
    if (!output_file.is_open()) {
        std::cerr << "Could not open output file '" << output_filename << "'!\n";
        return EXIT_FAILURE;
    }

    std::vector<Edge> edges;
    std::string buffer;

    int max_v = 0;
    while (input_file.good() && !input_file.eof()) {
        std::getline(input_file, buffer);
        if (!buffer.empty()) {
            auto const edge = parse_edge(buffer);
            max_v = std::max({max_v, edge.first, edge.second});
            edges.push_back(edge);
        }
    }

    if (input_file.bad()) {
        std::cerr << "Error while reading from file!\n";
        return EXIT_FAILURE;
    }

    count_cliques(edges, output_file, k, max_v);

    return EXIT_SUCCESS;
}