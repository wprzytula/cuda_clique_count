#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "common/errors.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <memory>
#include <vector>
#include <utility>
#include <charconv>
#include <algorithm>
#include <numeric>
#include <cassert>
#include <unordered_map>

#ifdef PRINT
#define debug(x) x
#else
#define debug(x)
#endif

#define CEIL_DIV(x, y) ((x + y - 1) / y)
#define MAX_K 12
#define MAX_DEG 1024
#define MODULO 1'000'000'000;

#define BLOCK_SIZE 128
// #define BLOCK_SIZE 32
#define NUM_BLOCKS 128
// #define NUM_BLOCKS 1

#define WARP_SIZE 32
#define WARPS_PER_BLOCK (BLOCK_SIZE / WARP_SIZE)
#define NUM_WARPS (NUM_BLOCKS * WARPS_PER_BLOCK)

namespace cpu { namespace {
    using Edge = std::pair<int, int>;

    // returns max_v
    int make_vertices_consecutive_natural_numbers(std::vector<Edge>& edges) {
        int next_num = 0;
        std::unordered_map<int, int> map;
        for (auto const [v1, v2]: edges) {
            if (map.find(v1) == map.cend()) {
                map[v1] = next_num++;
            }
            if (map.find(v2) == map.cend()) {
                map[v2] = next_num++;
            }
        }
        for (auto& edge: edges) {
            edge.first = map[edge.first];
            edge.second = map[edge.second];
        }
        return next_num - 1;
    }

    std::vector<int> compute_degs(std::vector<Edge> const& edges, int max_vertex) {
        std::vector<int> deg;
        deg.resize(max_vertex + 1);

        for (auto const [v1, v2]: edges) {
            ++deg[v1];
            ++deg[v2];
        }

        return deg;
    }

    void orient_graph(std::vector<Edge>& edges, std::vector<int> const& deg) {
        for (Edge& edge: edges) {
            auto [v1, v2] = edge;
            int const deg_v1 = deg[v1];
            int const deg_v2 = deg[v2];
            if (deg_v1 > deg_v2 || (deg_v1 == deg_v2 && v1 > v2)) {
                // Revert the edge
                edge.first = v2;
                edge.second = v1;
            }
        }
    }

    struct CSR {
        std::vector<int> col_idx;
        std::vector<int> row_ptr;
        int max_v;
        int n;

        CSR(std::vector<Edge> const& edges, int const max_v) : max_v{max_v}, n{max_v + 1} {
            assert(std::is_sorted(edges.cbegin(), edges.cend()));

            col_idx.resize(edges.size());
            row_ptr.resize(n + 1);

            int col_i = 0;
            for (int row = 0; row <= max_v; ++row) {
                row_ptr[row] = col_i;
                while (col_i < edges.size() && edges[col_i].first == row) {
                    // std::cerr << "Col_i: " << col_i << std::endl;
                    col_idx[col_i] = edges[col_i].second;
                    ++col_i;
                }
            }
            row_ptr[n] = col_idx.size();
        }
    };

debug(
    std::ostream& operator<<(std::ostream &os, CSR const& csr) {
        os << "Col_idx: [ ";
        for (int col: csr.col_idx) {
            os << col << ", ";
        }
        os << " ]\n";
        os << "Row_ptr: [ ";
        for (int row: csr.row_ptr) {
            os << row << ", ";
        }
        os << "]\n";

        return os;
    })

    Edge parse_edge(std::string const& buf) {
        char const* ptr = buf.data();
        int v1, v2;
        auto res1 = std::from_chars(ptr, ptr + buf.size(), v1);
        if (res1.ec != std::errc()) {
            std::cerr << "Error while parsing first vertex int!\n";
            std::cerr << "(problematic line: " << ptr << ")\n";
            exit(EXIT_FAILURE);
        }
        ptr = res1.ptr;
        while (std::isspace(*ptr)) ++ptr;

        auto res2 = std::from_chars(ptr, buf.data() + buf.size(), v2);
        if (res2.ec != std::errc()) {
            std::cerr << "Error while parsing second vertex int!\n";
            std::cerr << "(problematic line: " << ptr << ")\n";
            exit(EXIT_FAILURE);
        }
        return {v1, v2};
    }
}} // namespace


struct CSR {
    int vs;
    int* row_ptr;
    int row_len;
    int* col_idx;
    int col_len;
};

struct InducedSubgraph {
    int len_qwords;
    int vs;
    int mapping[MAX_DEG];
    unsigned long long adjacency_matrix[MAX_DEG * MAX_DEG / 64];

    __device__ void extract(CSR const& graph, int const v) {
        int const tid = threadIdx.x;
        int const start = graph.row_ptr[v];
        int const end = graph.row_ptr[v + 1];

        if (tid == 0) {
            vs = end - start;
            len_qwords = CEIL_DIV(vs, 64);
        }
        __syncthreads();

/* Build subgraph mapping: new_vertex [0..1024] -> old_vertex [0..|V|] */
        for (int j = tid; start + j < end; j += blockDim.x) {
            // put neighbours in mapping.
            int const neighbour = graph.col_idx[start + j];
            mapping[j] = neighbour;
        }

        __syncthreads();

/* Build adjacency matrix  */
        // It has k rows, where k = |induced subgraph vertices|
        auto const& mapping = this->mapping;
        auto old = [&mapping](int new_v){/* std::cout << "old(" << new_v << ")\n";  */return mapping[new_v];};
        auto neigh = [&graph](int col_i){return graph.col_idx[col_i];};

        // For each row
        for (int i = tid; i < vs; i += blockDim.x) {
            // Retrieve old id of the vertex
            int const old_v1 = mapping[i];

            // Operate on this row
            auto *const row = adjacency_matrix + i * vs;

            // Clear the row after previous subgraph (zeros are assumed in the algorithm)
            for (int j = 0; j < len_qwords; ++j) {
                row[j] = 0;
            }

            int csr_idx = graph.row_ptr[old_v1];
            int const csr_idx_end = graph.row_ptr[old_v1 + 1];

            // For each cell in this row
            for (int adj_idx = 0; adj_idx < vs; ++adj_idx) {
                // std::cout << "Incremented adj_idx to " << adj_idx << ", now points to " << old(adj_idx) << "\n";

                if (csr_idx >= csr_idx_end) {
                    // csr_idx went out of bounds.
                    goto end_row;
                }

                while (neigh(csr_idx) < old(adj_idx)) {
                    // std::cout << "Incremented csr_idx to " << csr_idx << "\n";
                    ++csr_idx;
                    if (csr_idx >= csr_idx_end) {
                        // csr_idx went out of bounds.
                        goto end_row;
                    }
                    // std::cout << "csr_idx now points to " << neigh(csr_idx) << "\n";
                }

                // printf("Deciding edge between %d and %d based on value in csr_idx under %d: %d\n",
                //      old_v1, old(adj_idx), csr_idx, neigh(csr_idx));
                row[adj_idx / 64] |= ((unsigned long long)(neigh(csr_idx) == old(adj_idx))) << (adj_idx % 64);
end_row:            ;
            }
        }
    }
};

__device__ void print_subgraph(InducedSubgraph const& subgraph) {
        printf("Subgraph mapping: [ ");
        for (int i = 0; i < subgraph.vs; ++i) {
            int const old_v = subgraph.mapping[i];
            printf("%i ", old_v);
        }
        printf("]\n");
        printf("Adjacency matrix:\n");
        printf("  ");
        for (int i = 0; i < subgraph.vs; ++i) {
            int const old_v = subgraph.mapping[i];
            printf("%i ", old_v);
        }
        printf("\n");
        for (int i = 0; i < subgraph.vs; ++i) {
            printf("[");
            for (int j = 0; j < subgraph.vs; ++j) {
                bool exists = subgraph.adjacency_matrix[i * subgraph.vs + j / 64] & (1ULL << j % 64);
                printf(" %c", exists ? 'x' : ' ');
            }
            printf(" ]\n");
        }
    }

struct Stack {
    // VertexSet
    unsigned long long *vertices; // 2-level array [[true, true, false], [false, false, false]]

    bool* done;

    int* level; // len: stack_entries_num
};

struct Data {
    int k;
    int* next_vertex;
    Stack stacks[NUM_WARPS];
    CSR csr;
    InducedSubgraph* subgraphs;

    void init(cpu::CSR const& edges, int const k) {
        this->k = k;
        csr.vs = edges.n;

        csr.row_len = edges.row_ptr.size();
        csr.row_ptr = nullptr;
        HANDLE_ERROR(hipMalloc(&csr.row_ptr, edges.row_ptr.size() * sizeof(int)));
        HANDLE_ERROR(hipMemcpy(
            csr.row_ptr,
            edges.row_ptr.data(),
            edges.row_ptr.size() * sizeof(int),
            hipMemcpyHostToDevice)
        );

        csr.col_len = edges.col_idx.size();
        csr.col_idx = nullptr;
        HANDLE_ERROR(hipMalloc(&csr.col_idx, edges.col_idx.size() * sizeof(int)));
        HANDLE_ERROR(hipMemcpy(
            csr.col_idx,
            edges.col_idx.data(),
            edges.col_idx.size() * sizeof(int),
            hipMemcpyHostToDevice)
        );

        subgraphs = nullptr;
        HANDLE_ERROR(hipMalloc(&subgraphs, NUM_BLOCKS * sizeof(InducedSubgraph)));

        int const storage_units_per_vertex_set = MAX_DEG / 64;
        // Initialise stacks
        int const max_entries = k * MAX_DEG / WARPS_PER_BLOCK;
        for (int i = 0; i < NUM_WARPS; ++i) {
            Stack& stack = stacks[i];
            stack.vertices = nullptr;
            HANDLE_ERROR(hipMalloc(&stack.vertices, max_entries * storage_units_per_vertex_set * sizeof(*stack.vertices)));
            HANDLE_ERROR(hipMemset(stack.vertices, -1 /*1*/, storage_units_per_vertex_set * sizeof(*stack.vertices))); // first stack entry

            stack.level = nullptr;
            HANDLE_ERROR(hipMalloc(&stack.level, max_entries * sizeof(*stack.level)));
            HANDLE_ERROR(hipMalloc(&stack.done, max_entries * sizeof(*stack.done)));
        }

        next_vertex = nullptr;
        HANDLE_ERROR(hipMalloc(&next_vertex, sizeof(*next_vertex)));
        HANDLE_ERROR(hipMemset(next_vertex, 0, sizeof(*next_vertex)));
    }
};

__constant__ Data global_data;

// https://stackoverflow.com/a/3208376
#define BYTE_TO_BINARY_PATTERN "%c%c%c%c%c%c%c%c"
#define BYTE_TO_BINARY(byte)  \
  ((byte) & 0x80 ? '1' : '0'), \
  ((byte) & 0x40 ? '1' : '0'), \
  ((byte) & 0x20 ? '1' : '0'), \
  ((byte) & 0x10 ? '1' : '0'), \
  ((byte) & 0x08 ? '1' : '0'), \
  ((byte) & 0x04 ? '1' : '0'), \
  ((byte) & 0x02 ? '1' : '0'), \
  ((byte) & 0x01 ? '1' : '0')

#define QWORD_TO_BINARY_HIGHER(name, qword) \
printf(name ": Bytes 7, 6, 5, 4: " BYTE_TO_BINARY_PATTERN " " BYTE_TO_BINARY_PATTERN " " BYTE_TO_BINARY_PATTERN " " BYTE_TO_BINARY_PATTERN "\n",\
    BYTE_TO_BINARY(qword>>56), BYTE_TO_BINARY(qword>>48), BYTE_TO_BINARY(qword>>40), BYTE_TO_BINARY(qword>>32));

#define QWORD_TO_BINARY_LOWER(name, qword) \
printf(name ": Bytes 3, 2, 1, 0: " BYTE_TO_BINARY_PATTERN " " BYTE_TO_BINARY_PATTERN " " BYTE_TO_BINARY_PATTERN " " BYTE_TO_BINARY_PATTERN "\n",\
    BYTE_TO_BINARY(qword>>24), BYTE_TO_BINARY(qword>>16), BYTE_TO_BINARY(qword>>8), BYTE_TO_BINARY(qword));

__device__ void intersect_adjacent(InducedSubgraph const& subgraph, unsigned long long const* vertex_set, int vertex, unsigned long long* out_vertex_set) {
    auto const* row = subgraph.adjacency_matrix + vertex * subgraph.vs;

    for (int i = threadIdx.x % WARP_SIZE; i < subgraph.len_qwords; i += WARP_SIZE) {
        debug({
            printf("Block %i, Thread %i: I'm intersecting %i-th vertex slice: vertex_set[%i]=(%lli), row[%i] = (%llx)\n",
                blockIdx.x, threadIdx.x, i, i, vertex_set[i], i, row[i]);
            QWORD_TO_BINARY_HIGHER("Set", vertex_set[i]);
            QWORD_TO_BINARY_LOWER("Set", vertex_set[i]);
            QWORD_TO_BINARY_HIGHER("Row", row[i]);
            QWORD_TO_BINARY_LOWER("Row", row[i]);
        });
        out_vertex_set[i] = vertex_set[i] & row[i]; // set each vertex as in or out of set
    }
}

__device__ void copy_adjacent(InducedSubgraph const& subgraph, int vertex, unsigned long long* out_vertex_set) {
    auto const* row = subgraph.adjacency_matrix + vertex * subgraph.vs;

    for (int i = threadIdx.x; i < subgraph.len_qwords; i += WARP_SIZE) {
        debug({
            printf("Block %i, Thread %i: I'm copying %i-th vertex slice: row[%i] = (%llx)\n",
                blockIdx.x, threadIdx.x, i, i, row[i]);
            QWORD_TO_BINARY_HIGHER("Row", row[i]);
            QWORD_TO_BINARY_LOWER("Row", row[i]);
        });
        out_vertex_set[i] = row[i]; // set each vertex as in or out of row
    }
}

__device__ bool vertex_set_nonempty(unsigned long long const* set, int const vs) {
    int const tid = threadIdx.x % WARP_SIZE;
    int const warp_id = threadIdx.x / WARP_SIZE;

    int const len_qwords = CEIL_DIV(vs, 64);

    __shared__ bool nonempty[WARPS_PER_BLOCK];

    // for (int i = tid; i < len_qwords; i += WARP_SIZE) {
    //     if (i + 1 == len_qwords) { // if last, we have to only take into account the valid bits.

    //         // vs % 64
    //         // 0 -> 1 1 ... 1 1 1
    //         // 1 -> 0 0 ... 0 0 1
    //         // 2 -> 0 0 ... 0 1 1
    //         // ...
    //         // 63 -> 0 1 ... 1 1 1

    //         unsigned long long const mask = (-1ULL) >> ((64 - vs % 64) % 64);
    //         nonempty |= set[i] & mask;
    //     } else {
    //         nonempty |= set[i];
    //     }
    // }

    if (tid == 0) {
        for (int i = 0; i < len_qwords; ++i) {
            nonempty[warp_id] |= set[i];
        }
    }
    __syncwarp();

    debug(
        if (tid == 0) {
            printf("set_nonempty([ ");
            for (int i = 0; i < vs; ++i) {
                printf("%i ", !!(set[i / 64] & (1ULL << i % 64)));
            }
            printf("]) = %i\n", nonempty[warp_id]);
        }
    )

    return nonempty[warp_id];
}

__device__ int acquire_next_vertex(Data const& data) {
    int const thread_id = threadIdx.x;
    __shared__ int chosen_vertex;

    if (thread_id == 0) {
        chosen_vertex = atomicAdd(data.next_vertex, 1);
        // printf("Block %i: Acquired vertex %i.\n", blockIdx.x, chosen_vertex);
    }
    __syncthreads();
    return chosen_vertex;
}

__device__ bool vertex_set_contains(unsigned long long const* vertex_set, int const current_frame, int const v) {
    // if (debug && threadIdx.x == 0)
    //     printf("Thread %i: set: %p, current: %i, v: %i\n", threadIdx.x, vertex_set, current_frame, v);
    // __syncthreads();

    return vertex_set[MAX_DEG / 64 * current_frame + v / 64] & (1ULL << (v % 64));
}

// Graph traversal for graph orientation method
// 1 𝑛𝑢𝑚𝐶𝑙𝑖𝑞𝑢𝑒𝑠 = 0
// 2 procedure 𝑡𝑟𝑎𝑣𝑒𝑟𝑠𝑒𝑆𝑢𝑏𝑡𝑟𝑒𝑒 (𝐺, 𝑘, ℓ, 𝐼 ) : (G: Graph, k: clique_size, l: current_level, I: set_of_vertices)
// 3 for 𝑣 ∈ 𝐼
// 4    𝐼 ′ = 𝐼 ∩ 𝐴𝑑𝑗_𝐺 (𝑣)
// 5    if ℓ + 1 == 𝑘
// 6        𝑛𝑢𝑚𝐶𝑙𝑖𝑞𝑢𝑒𝑠 + = |𝐼 ′ |
// 7    else if |𝐼 ′ | > 0
// 8        𝑡𝑟𝑎𝑣𝑒𝑟𝑠𝑒𝑆𝑢𝑏𝑡𝑟𝑒𝑒 (𝐺, 𝑘, ℓ + 1, 𝐼 ′ )
__global__ void kernel(unsigned long long *count) {
    int const block_id = blockIdx.x;
    int const thread_id = threadIdx.x % WARP_SIZE;
    int const warp_id = threadIdx.x / WARP_SIZE;
    int const unique_warp_id = block_id * WARPS_PER_BLOCK + warp_id;

    int chosen_vertex;
    Data& data = global_data;
    int const all_vs = data.csr.vs;

    Stack& stack = data.stacks[unique_warp_id];
    __shared__ int stack_tops[WARPS_PER_BLOCK];
    int& stack_top = stack_tops[warp_id];

    debug(if (block_id == 0 && warp_id == 0 && thread_id == 0) printf("\n\n----- RUNNING KERNEL!!! ------\n\n"));

    __shared__ int cliques[WARPS_PER_BLOCK][MAX_K];
    // Set counters to zeros.
    if (thread_id < MAX_K) {
        cliques[warp_id][thread_id] = 0;
    }

    __syncthreads();

    while ((chosen_vertex = acquire_next_vertex(data)) < all_vs) {
        debug(if (thread_id == 0 && warp_id == 0) {
            printf("\n ACQUISITION: Block %i has acquired vertex %i\n", block_id, chosen_vertex);
        })

        // Compute InducedSubgraph
        {
            InducedSubgraph& subgraph = data.subgraphs[block_id];
            subgraph.extract(data.csr, chosen_vertex);
            debug(if (thread_id == 0 && warp_id == 0) print_subgraph(subgraph));
        }
        InducedSubgraph const& subgraph = data.subgraphs[block_id];
        int const vs = subgraph.vs;

        // Initialise empty stack.
        if (thread_id == 0) {
            stack_top = -1;
        }

        // First level
        __syncthreads();
        if (warp_id == 0) {
            // We've found a number=vs level 1 cliques.
            if (thread_id == 0) {
                debug(printf("Block %i vertex %i: found %i of 2-cliques.\n",
                             block_id, chosen_vertex, vs);
                );
                int* level_cliques = &cliques[0/*warp id*/][1 /*level*/];
                *level_cliques = (*level_cliques + vs) % MODULO;
            }
            if (2 < data.k) { // entry.level + 1 < k
                for (int v = 0; v < vs; ++v) {
                    int const assigned_warp = v % WARPS_PER_BLOCK;
                    int const unique_assigned_warp = block_id * WARPS_PER_BLOCK + assigned_warp;
                    int& assigned_stack_top = stack_tops[assigned_warp];
                    Stack& assigned_stack = data.stacks[unique_assigned_warp];
                    debug(if (thread_id == 0) printf("Block %i, Vertex %i, Assigned subgraph's vertex %i to warp %i.\n",
                                                        block_id, chosen_vertex, v, assigned_warp)
                    );
                    __syncwarp();

                    unsigned long long* new_vertices = assigned_stack.vertices + (assigned_stack_top + 1) * MAX_DEG / 64;
                    debug(if (thread_id == 0) printf("Block %i, Vertex %i, Warp %i: Copying subgraph's vertex %i.\n",
                                                        block_id, chosen_vertex, warp_id, v)
                    );
                    copy_adjacent(subgraph, v, new_vertices);

                    __syncwarp();

                    if (vertex_set_nonempty(new_vertices, vs)) {
                        // stack.emplace(new_vertices, entry.level + 1);
                        if (thread_id == 0) {
                            ++assigned_stack_top;
                            assigned_stack.level[assigned_stack_top] = 2;
                            assigned_stack.done[assigned_stack_top] = false;
                        }
                    }
                    __syncwarp();
                }
            }
        }

        __syncthreads();
        // Per-warp iteration
        debug(if (warp_id == 0 && thread_id == 0)
            printf("\n BEGINNING PER_WARP iteration: block %i, vertex %i.\n\n", block_id, chosen_vertex)
        );

        while (stack_top >= 0) {
            __syncwarp();
            int const current = stack_top;
            debug(if (thread_id == 0) printf("Warp %i: Stack top: %i\n", warp_id, current));
            debug(if (thread_id == 0) {
                printf("Block %i warp %i vertex %i operating on stack entry with idx %i, done? %i\n",
                        block_id, warp_id, chosen_vertex, current, stack.done[current]);
            })
            if (stack.done[current]) {
                if (thread_id == 0)
                    --stack_top;
                __syncwarp();
                continue;
            }
            for (int v = 0; v < vs; ++v) {
                __syncwarp();
                if (vertex_set_contains(stack.vertices, current, v)) { // entry.vertices.contains(v)
                    // We've found a `level`-level clique.
                    if (thread_id == 0) {
                        int* level_cliques = &cliques[warp_id][stack.level[current]];
                        *level_cliques = (*level_cliques + 1) % MODULO;
                        debug(
                            printf("Block %i, warp %i, vertex %i: found a %i-clique. Cliques now: %i\n",
                                      block_id, warp_id, chosen_vertex, stack.level[current] + 1, *level_cliques)
                        );
                    }

                    // Let's explore deeper.
                    if (stack.level[current] + 1 < data.k) { // entry.level + 1 < k
                        unsigned long long* new_vertices = stack.vertices + (stack_top + 1) * MAX_DEG / 64;
                        debug(if (thread_id == 0) printf("Block %i, Vertex %i, Warp %i: Intersecting with subgraph's vertex %i.\n",
                                                         block_id, chosen_vertex, warp_id, v)
                        );
                        intersect_adjacent(subgraph, stack.vertices + current * MAX_DEG / 64, v, new_vertices);

                        __syncwarp();

                        if (vertex_set_nonempty(new_vertices, vs)) {
                            // stack.emplace(new_vertices, entry.level + 1);
                            if (thread_id == 0) {
                                ++stack_top;
                                stack.level[stack_top] = stack.level[current] + 1;
                                stack.done[stack_top] = false;
                            }
                        }
                    }
                }
            }
            __syncwarp();

            if (thread_id == 0) {
                stack.done[current] = true;
                if (current == stack_top) /*leaf reached, go back*/{
                    debug(printf("Vertex %i warp %i: Reached leaf in entry %i.\n", chosen_vertex, warp_id, current));
                    --stack_top;
                } else {
                    debug(printf("Vertex %i warp %i: Finished work over node in entry %i.\n", chosen_vertex, warp_id, current));
                }
            }

            __syncwarp();
        }
        debug(if (thread_id == 0) {
            printf("Block %i, Vertex %i, warp %i: Finished stack iteration.\n", block_id, chosen_vertex, warp_id);
        });
    }

    __syncthreads();

    // if (thread_id == 0) { // DEBUG@@@R#@@#@T@@$T$@T@
    //     debug(printf("block %i warp %i: 2-Cliques num=%i\n", block_id, warp_id, cliques[0][1]));
    // }

    // if (warp_id != 0) {
    //     if (thread_id < data.k) {
    //         atomicAdd_block(&cliques[0][thread_id], cliques[warp_id][thread_id]);
    //     }
    // }

    __syncthreads();

    if (thread_id < data.k) {
        atomicAdd(&count[thread_id], (unsigned long long)cliques[warp_id][thread_id]);
    }
    // for (int i = thread_id; i < data.k; i += blockDim.x) {
    //     atomicAdd(&count[i], (unsigned long long)cliques[warp_id][i]);
    // }

    debug(
        __syncthreads();
        if (thread_id == 0 && warp_id == 0) printf("Block %i, Finished!\n", block_id);
    );
}

static void count_cliques(std::vector<cpu::Edge>& edges, std::ofstream& output_file, int k) {
    debug({
        std::cout << "unoriented sorted edges before making vertices consecutive:\n";
        for (auto const [v1, v2]: edges) {
            std::cout << "(" << v1 << ", " << v2 << ")\n";
        }
    })

    int const max_v = cpu::make_vertices_consecutive_natural_numbers(edges);

    debug({
        std::cout << "unoriented sorted edges with vertices made consecutive:\n";
        for (auto const [v1, v2]: edges) {
            std::cout << "(" << v1 << ", " << v2 << ")\n";
        }
        std::cout << "max_v=" << max_v << ")\n";
    });

    std::sort(edges.begin(), edges.end());
    debug({
        cpu::CSR unoriented_graph(edges, max_v);
        std::cout << "unoriented graph:\n";
        std::cout << unoriented_graph << "\n";
    });

    auto degs = cpu::compute_degs(edges, max_v);
    cpu::orient_graph(edges, degs);
    std::sort(edges.begin(), edges.end());

    debug({
        std::cout << "oriented sorted edges:\n";
        for (auto const [v1, v2]: edges) {
            std::cout << "(" << v1 << ", " << v2 << ")\n";
        }
    });

    cpu::CSR graph{edges, max_v};
    debug({
        std::cout << "oriented graph:\n";
        std::cout << graph << "\n";
    });

    auto cliques_cpu = std::make_unique<unsigned long long[]>(k);

    { // GPU section
        // input data
        Data data;
        data.init(graph, k);

        hipMemcpyToSymbol(HIP_SYMBOL(global_data), &data, sizeof(Data), 0, hipMemcpyHostToDevice);

        // output data
        unsigned long long *cliques_gpu;

        HANDLE_ERROR(hipMalloc(&cliques_gpu, k * sizeof(*cliques_gpu)));
        HANDLE_ERROR(hipMemset(cliques_gpu, 0, k * sizeof(*cliques_gpu)));

        hipEvent_t kernel_run, stop;
        hipEventCreate(&kernel_run);
        hipEventCreate(&stop);

        hipEventRecord(kernel_run, 0);

        // RUN KERNEL, RUN!
        kernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(cliques_gpu);


        // Get back the output data
        HANDLE_ERROR(hipMemcpy(cliques_cpu.get(),
                cliques_gpu,
                k * sizeof(*cliques_gpu),
                hipMemcpyDeviceToHost)
        );

        HANDLE_ERROR(hipFree(cliques_gpu));

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float elapsed_kernel;
        HANDLE_ERROR(hipEventElapsedTime(&elapsed_kernel, kernel_run, stop));
        printf("Elapsed kernel: %.3fms\n", elapsed_kernel);

        hipEventDestroy(kernel_run);
        hipEventDestroy(stop);
    }


    cliques_cpu[0] = max_v + 1;
    for (int i = 1; i < k; ++i) {
        cliques_cpu[i] = cliques_cpu[i] % MODULO;
    }

    std::stringstream s;

    s << "count: [ ";
    output_file << cliques_cpu[0];
    s << cliques_cpu[0];
    for (int i = 1; i < k; ++i) {
        output_file << ' ' << cliques_cpu[i];
        s << ' ' << cliques_cpu[i];
    }
    s << " ]\n";
    // if (debug)
    std::cout << s.str();
}

int main(int argc, char const* argv[]) {
    if (argc != 4) {
        fprintf(stderr, "Bad arg num (expected 4, got %d)\n", argc);
        return EXIT_FAILURE;
    }

    char const* input_filename = argv[1];
    char const* k_str = argv[2];
    char const* output_filename = argv[3];

    std::ifstream input_file{input_filename, std::ios::in};
    if (!input_file.is_open()) {
        std::cerr << "Could not open input file '" << input_filename << "'!\n";
        return EXIT_FAILURE;
    }

    int k;
    try {
        k = std::stoi(k_str);
    } catch (std::invalid_argument&) {
        std::cerr << "Non integer k: " << k_str << "'!\n";
        return EXIT_FAILURE;
    } catch (std::out_of_range&) {
        std::cerr << "k too big for int type: " << k_str << "'!\n";
        return EXIT_FAILURE;
    }

    std::ofstream output_file{output_filename, std::ios::out};
    if (!output_file.is_open()) {
        std::cerr << "Could not open output file '" << output_filename << "'!\n";
        return EXIT_FAILURE;
    }

    std::vector<cpu::Edge> edges;
    std::string buffer;

    while (input_file.good() && !input_file.eof()) {
        std::getline(input_file, buffer);
        if (!buffer.empty()) {
            auto const edge = cpu::parse_edge(buffer);
            edges.push_back(edge);
        }
    }

    if (input_file.bad()) {
        std::cerr << "Error while reading from file!\n";
        return EXIT_FAILURE;
    }

    count_cliques(edges, output_file, k);

    return EXIT_SUCCESS;
}